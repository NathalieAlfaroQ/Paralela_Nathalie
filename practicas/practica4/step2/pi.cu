// Written by Barry Wilkinson, UNC-Charlotte. Pi.cu December 22, 2010.
// Derived somewhat from code developed by Patrick Rogers, UNC-C

// Para correr CUDA sin NVIDIA, seguir tutorial 
// https://colab.research.google.com/drive/1GJOfTp56OeQRdE4u2_S7pUNRcJb4ik9X?usp=sharing#scrollTo=4UgGjl7LSBeG

// Compilar nvcc -o pi pi.cu
// Ejecuta ./pi

/*
RESULTADO AL EJECUTAR:

lapz@LapzDesktop:~/nath$ nvcc -o pi pi.cu
lapz@LapzDesktop:~/nath$ ./pi

# of trials per thread = 4096, # of blocks = 256, # of threads/block = 256.

GPU pi calculated in 0.170903 s.

CPU pi calculated in 2.787307 s.

CUDA estimate of PI = 3.141582 [error of -0.000011]
CPU estimate of PI = 3.141581 [error of -0.000012]
*/

// Bibliotecas
#include <stdlib.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 
#include <math.h> 
#include <time.h> 
#include <hiprand/hiprand_kernel.h> 
 
// Constantes
#define PI 3.1415926535
#define TRIALS_PER_THREAD 4096 
#define BLOCKS 256 
#define THREADS 256 

/*
Realiza el cálculo de Monte Carlo en la GPU:
Inicializa el estado del generador de números aleatorios para cada hilo.
Genera puntos aleatorios en el cuadrado unitario (0,0) a (1,1).
Cuenta cuántos puntos caen dentro del círculo unitario de radio 1.
Calcula la estimación de π basada en la proporción de puntos dentro del círculo y
almacena el resultado en la memoria global.
*/
__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {

    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x; 
    int points_in_circle = 0; 
    float x, y; 
    hiprand_init(1234, tid, 0, &states[tid]);

    for(int i = 0; i < TRIALS_PER_THREAD; i++) { 
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        points_in_circle += (x*x + y*y <= 1.0f); 
    } // End for

    estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; 
} // End gpu_monte_carlo

/*
Genera puntos aleatorios en el cuadrado unitario (0,0) a (1,1).
Cuenta cuántos puntos caen dentro del círculo unitario de radio 1.
Calcula la estimación de π basada en la proporción de puntos dentro del círculo.
*/
float host_monte_carlo(long trials) { 
    float x, y; 
    long points_in_circle = 0; 

    for(long i = 0; i < trials; i++) {
        x = rand() / (float) RAND_MAX; 
        y = rand() / (float) RAND_MAX; 
        points_in_circle += (x*x + y*y <= 1.0f); 
    } // End for

    return 4.0f * points_in_circle / trials;
} // End host_monte_carlo

/*
Configura los parámetros de CUDA y asigna memoria en la GPU.
Llama a la función gpu_monte_carlo para realizar el cálculo en la GPU.
Transfiere los resultados de la GPU a la CPU y calcula la estimación final de π.
Mide y muestra el tiempo de cálculo en la GPU.
Llama a la función host_monte_carlo para realizar el cálculo en la CPU.
Mide y muestra el tiempo de cálculo en la CPU.
Compara y muestra las estimaciones de π y sus errores respecto al valor conocido.
Libera la memoria asignada en la GPU.
*/
int main (int argc, char *argv[]) {

    clock_t start, stop; 
    float host[BLOCKS * THREADS]; 
    float *dev; 
    hiprandState *devStates; 
 
    printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", TRIALS_PER_THREAD, BLOCKS, THREADS); 
 
    start = clock(); 
 
    hipMalloc((void **)&dev, BLOCKS * THREADS * sizeof(float));
    
    hipMalloc((void **)&devStates, BLOCKS * THREADS * sizeof(hiprandState));
    
    gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates); 

    hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost);

    float pi_gpu = 0;

    for(int i = 0; i < BLOCKS * THREADS; i++) { 
        pi_gpu += host[i]; 
    } // End for
 
    pi_gpu /= (BLOCKS * THREADS); 
 
    stop = clock(); 
 
    printf("\nGPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC); 
 
    start = clock(); 

    float pi_cpu = host_monte_carlo(BLOCKS * THREADS * TRIALS_PER_THREAD); 

    stop = clock();

    printf("\nCPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC); 
 
    printf("\nCUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI); 

    printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI); 
  
    hipFree(dev);
    hipFree(devStates);

    // End program
    return 0; 
} // End main