// g++ -o step1 step1.cpp
// ./step1
// Max error: 0.000000

/*
nvcc step1.cu -o step1
> ./step1
Max error: 0.000000

https://developer.nvidia.com/blog/even-easier-introduction-cuda/

nvprof ./step1

==12739== NVPROF is profiling process 12739, command: ./step1
Max error: 0
==12739== Profiling application: ./step1
==12739== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:  100.00%  180.44ms         1  180.44ms  180.44ms  180.44ms  add(int, float*, float*)
      API calls:   51.19%  189.91ms         2  94.956ms  43.217us  189.87ms  cudaMallocManaged
                   48.65%  180.48ms         1  180.48ms  180.48ms  180.48ms  cudaDeviceSynchronize
                    0.10%  378.76us         2  189.38us  173.49us  205.27us  cudaFree
                    0.04%  159.70us       101  1.5810us     204ns  62.573us  cuDeviceGetAttribute
                    0.01%  38.291us         1  38.291us  38.291us  38.291us  cudaLaunchKernel
                    0.00%  17.037us         1  17.037us  17.037us  17.037us  cuDeviceGetName
                    0.00%  10.609us         1  10.609us  10.609us  10.609us  cuDeviceGetPCIBusId
                    0.00%  8.2550us         1  8.2550us  8.2550us  8.2550us  cuDeviceTotalMem
                    0.00%  1.9300us         3     643ns     306ns  1.2870us  cuDeviceGetCount
                    0.00%  1.2280us         2     614ns     269ns     959ns  cuDeviceGet
                    0.00%     457ns         1     457ns     457ns     457ns  cuDeviceGetUuid

==12739== Unified Memory profiling result:
Device "NVIDIA GeForce GTX 1060 3GB (0)"
   Count  Avg Size  Min Size  Max Size  Total Size  Total Time  Name
      48  170.67KB  4.0000KB  0.9961MB  8.000000MB  737.0930us  Host To Device
      24  170.67KB  4.0000KB  0.9961MB  4.000000MB  335.4710us  Device To Host
      12         -         -         -           -  2.956630ms  Gpu page fault groups
Total CPU Page faults: 36
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays on the GPU

// function to add the elements of two arrays
/*__global__ void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}*/

__global__ void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  // 2**20
  int N = 1 << 20; // 1M elements

  // Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  // float *x = new float[N];
  // float *y = new float[N];

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  // add(N, x, y);
  // add<<<1, 1>>>(N, x, y);
  add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  // delete [] x;
  // delete [] y;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}