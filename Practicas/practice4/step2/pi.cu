// Written by Barry Wilkinson, UNC-Charlotte. Pi.cu December 22, 2010.
// Derived somewhat from code developed by Patrick Rogers, UNC-C

// Para correr CUDA sin NVIDIA, seguir tutorial 
// https://colab.research.google.com/drive/1GJOfTp56OeQRdE4u2_S7pUNRcJb4ik9X?usp=sharing#scrollTo=4UgGjl7LSBeG

// Compilar nvcc -o pi pi.cu
// Ejecuta ./pi

/*
RESULTADO AL EJECUTAR:

lapz@LapzDesktop:~/nath$ nvcc -o pi pi.cu
lapz@LapzDesktop:~/nath$ ./pi

# of trials per thread = 4096, # of blocks = 256, # of threads/block = 256.

GPU pi calculated in 0.170903 s.

CPU pi calculated in 2.787307 s.

CUDA estimate of PI = 3.141582 [error of -0.000011]
CPU estimate of PI = 3.141581 [error of -0.000012]
*/

#include <stdlib.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 
#include <math.h> 
#include <time.h> 
#include <hiprand/hiprand_kernel.h> 
 
#define PI 3.1415926535
#define TRIALS_PER_THREAD 4096 
#define BLOCKS 256 
#define THREADS 256 
 
__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {

    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x; 
    int points_in_circle = 0; 
    float x, y; 

    hiprand_init(1234, tid, 0, &states[tid]);

    for(int i = 0; i < TRIALS_PER_THREAD; i++) { 
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        points_in_circle += (x*x + y*y <= 1.0f); 
    }

    estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; 
}
 
float host_monte_carlo(long trials) { 
    float x, y; 
    long points_in_circle = 0; 

    for(long i = 0; i < trials; i++) {
        x = rand() / (float) RAND_MAX; 
        y = rand() / (float) RAND_MAX; 
        points_in_circle += (x*x + y*y <= 1.0f); 
    } 

    return 4.0f * points_in_circle / trials;
} 
 
int main (int argc, char *argv[]) {

    clock_t start, stop; 
    float host[BLOCKS * THREADS]; 
    float *dev; 
    hiprandState *devStates; 
 
    printf("# of trials per thread = %d, # of blocks = %d, # of threads/block = %d.\n", TRIALS_PER_THREAD, BLOCKS, THREADS); 
 
    start = clock(); 
 
    hipMalloc((void **)&dev, BLOCKS * THREADS * sizeof(float));
    
    hipMalloc((void **)&devStates, BLOCKS * THREADS * sizeof(hiprandState));
    
    gpu_monte_carlo<<<BLOCKS, THREADS>>>(dev, devStates); 

    hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(float), hipMemcpyDeviceToHost);

    float pi_gpu = 0;

    for(int i = 0; i < BLOCKS * THREADS; i++) { 
        pi_gpu += host[i]; 
    } 
 
    pi_gpu /= (BLOCKS * THREADS); 
 
    stop = clock(); 
 
    printf("\nGPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC); 
 
    start = clock(); 

    float pi_cpu = host_monte_carlo(BLOCKS * THREADS * TRIALS_PER_THREAD); 

    stop = clock();

    printf("\nCPU pi calculated in %f s.\n", (stop-start)/(float)CLOCKS_PER_SEC); 
 
    printf("\nCUDA estimate of PI = %f [error of %f]\n", pi_gpu, pi_gpu - PI); 

    printf("CPU estimate of PI = %f [error of %f]\n", pi_cpu, pi_cpu - PI); 
  
    hipFree(dev);
    hipFree(devStates);

    return 0; 
}